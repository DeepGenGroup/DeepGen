#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "utils.h"

template <typename T>
__inline__ __device__ T warpReduceSum(T val) {
#pragma unroll
  for (int mask = 16; mask > 0; mask >>= 1) val += __shfl_xor_sync(uint32_t(-1), val, mask);
  return val;
}

/* Calculate the sum of all elements in a block */
template <typename T>
__inline__ __device__ T blockReduceSum(T val) {
  static __shared__ T shared[32];
  int lane = threadIdx.x & 0x1f;
  int wid = threadIdx.x >> 5;

  val = warpReduceSum<T>(val);

  if (lane == 0) shared[wid] = val;

  __syncthreads();

  // Modify from blockDim.x << 5 to blockDim.x / 32. to prevent
  // blockDim.x is not divided by 32
  val = (threadIdx.x < (blockDim.x / 32.f)) ? shared[lane] : (T)(0.0f);
  val = warpReduceSum<T>(val);
  return val;
}

// TODO(woosuk): Further optimize this kernel.
template <typename scalar_t>
__global__ void rms_norm_kernel(scalar_t* __restrict__ out,           // [..., hidden_size]
                                const scalar_t* __restrict__ input,   // [..., hidden_size]
                                const scalar_t* __restrict__ weight,  // [hidden_size]
                                const float epsilon, const int num_tokens, const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    const float x = (float)input[blockIdx.x * hidden_size + idx];
    variance += x * x;
  }
  variance = blockReduceSum<float>(variance);
  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float)input[blockIdx.x * hidden_size + idx];
    out[blockIdx.x * hidden_size + idx] = ((scalar_t)(x * s_variance)) * weight[idx];
  }
}

// TODO: Further optimize this kernel.
template <typename scalar_t>
__global__ void fused_add_rms_norm_kernel(scalar_t* __restrict__ input,         // [..., hidden_size]
                                          scalar_t* __restrict__ residual,      // [..., hidden_size]
                                          const scalar_t* __restrict__ weight,  // [hidden_size]
                                          const float epsilon, const int num_tokens, const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float)input[blockIdx.x * hidden_size + idx];
    x += (float)residual[blockIdx.x * hidden_size + idx];
    variance += x * x;
    residual[blockIdx.x * hidden_size + idx] = (scalar_t)x;
  }
  variance = blockReduceSum<float>(variance);
  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float)residual[blockIdx.x * hidden_size + idx];
    input[blockIdx.x * hidden_size + idx] = ((scalar_t)(x * s_variance)) * weight[idx];
  }
}

void rms_norm(torch::Tensor& out,     // [..., hidden_size]
              torch::Tensor& input,   // [..., hidden_size]
              torch::Tensor& weight,  // [hidden_size]
              float epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  DISPATCH_FLOATING_TYPES(input.scalar_type(), "rms_norm_kernel", [&] {
    rms_norm_kernel<scalar_t><<<grid, block, 0, stream>>>(out.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(), weight.data_ptr<scalar_t>(), epsilon, num_tokens, hidden_size);
  });
}

void fused_add_rms_norm(torch::Tensor& input,     // [..., hidden_size]
                        torch::Tensor& residual,  // [..., hidden_size]
                        torch::Tensor& weight,    // [hidden_size]
                        float epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  DISPATCH_FLOATING_TYPES(input.scalar_type(), "fused_add_rms_norm_kernel", [&] {
    fused_add_rms_norm_kernel<scalar_t><<<grid, block, 0, stream>>>(input.data_ptr<scalar_t>(), residual.data_ptr<scalar_t>(), weight.data_ptr<scalar_t>(), epsilon, num_tokens, hidden_size);
  });
}
