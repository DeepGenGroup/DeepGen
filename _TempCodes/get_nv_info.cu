#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    
    if (deviceCount == 0) {
        printf("No CUDA-capable devices found\n");
        return 1;
    }

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        
        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
        printf("  Compute Capability: %d.%d\n", 
               deviceProp.major, deviceProp.minor);
        printf("  Shared Memory Per SM (bytes): %zu\n", 
               deviceProp.sharedMemPerMultiprocessor);
        printf("  Shared Memory Per Block (bytes): %zu\n", 
               deviceProp.sharedMemPerBlock);
        
        // 获取最大动态共享内存大小（如果设备支持）
        int max_dynamic_shared_size;
        hipDeviceGetAttribute(&max_dynamic_shared_size, 
                              hipDeviceAttributeSharedMemPerBlockOptin, 
                              dev);
        printf("  Max Dynamic Shared Memory Per Block (bytes): %d\n", 
               max_dynamic_shared_size);
    }
    
    return 0;
}
