#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/driver_types.h>
#include <algorithm>
#include <vector>
#include <hip/hiprtc.h>
#include <stdio.h>
#include <fstream>
#include <sstream>
#include <hip/hip_runtime.h>

template <typename T>
void display(T *host, int len) {
    // 打印
    int mid = len / 2;
    int start = (rand() % (mid - 1)) + 1;
    int end = (rand() % (mid - 1)) + mid + 1;
    std::cout << "{" << host[0] << ", ..., " << host[start] << ", ..., "  << host[mid] << ", ..., "  << host[end] << ", ..., " << host[len - 1] << "}\n";
}

// cublas gemm
hipblasStatus_t cublasMatMulTransA(hipblasHandle_t handle, const float* A, const float* B, float* C, int M, int N, int K, bool isTranA, bool isTranB) {
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasStatus_t status;
    bool handleCreated = false;
    if (!handle) {
        status = hipblasCreate(&handle);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            return status;
        }
        handleCreated = true;
    }
    hipblasOperation_t tranA, tranB;
    int lda, ldb;
    if (isTranA && isTranB) {
        tranA = HIPBLAS_OP_N; tranB = HIPBLAS_OP_N;
        lda = M; ldb = K;
    } else if (!isTranA && isTranB) {
        tranA = HIPBLAS_OP_T; tranB = HIPBLAS_OP_N;
        lda = K; ldb = K;
    } else if (isTranA && !isTranB) {
        tranA = HIPBLAS_OP_N; tranB = HIPBLAS_OP_T;
        lda = M; ldb = N;
    } else {
        tranA = HIPBLAS_OP_T; tranB = HIPBLAS_OP_T;
        lda = K; ldb = N;
    }
    status = hipblasSgemm(handle,
                        tranA,   // A不转置
                        tranB,   // B转置
                        M,             // 结果矩阵行数
                        N,             // 结果矩阵列数
                        K,             // 公共维度
                        &alpha,
                        A, lda,          // A的维度M×K，lda=M
                        B, ldb,          // B的维度NxK，ldb=N
                        &beta,
                        C, M);         // C的维度M×N，ldc=M

    // 清理临时创建的句柄
    if (handleCreated) {
        hipblasDestroy(handle);
    }
    return status;
}

// gpu 验证
__global__ void verify_kernel(float* C, float* D, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m && col < n) {
      float sub = C[row * n + col] - D[col * m + row];
      if (sub >= 0.0000001f || sub <= -0.0000001f) {
        // printf("%d %d\n", row, col);
        printf("error!\nindex: (y=%d, x=%d)\nmine: %f  verify: %.8f\nsub: %.8f\n", row, col, C[row * n + col], D[col * m + row], sub);
      }
    }
}

// nvcc -o ./bin/runkernel runkernel.cu -lcuda -lcublas -arch=sm_80
int main(int argc, char** argv) {
    if (argc <= 8){
        std::cout << "Usage : M N K gridDims blockDims shmBytes cubinPath cubinFunc" << std::endl;
        return 1;
    }
    int device_count;
    hipError_t cuResult = hipInit(0);
    hipGetDeviceCount(&device_count);
    if (device_count == 0) {
        std::cerr << "没有找到可用的CUDA设备" << std::endl;
        return 1;
    }
    int device_id;
    hipGetDevice(&device_id);
    hipSetDevice(device_id);

    hipCtx_t context;
    cuResult = hipCtxCreate(&context, 0, device_id);
    if (cuResult != hipSuccess) {
        std::cerr << "Failed to create CUDA context: " << cuResult << std::endl;
        return 1;
    }

    // 加载.cubin模块
    hipModule_t module;
    hipError_t result = hipModuleLoad(&module, argv[7]);
    if (result != hipSuccess) {
        std::cerr << "Failed to load module: " << result << std::endl;
        return -1;
    }

    // 获取内核函数句柄
    hipFunction_t kernel;
    result = hipModuleGetFunction(&kernel, module, argv[8]);
    if (result != hipSuccess) {
        std::cerr << "Failed to get kernel function" << std::endl;
        return -1;
    }

    int M = std::stoi(argv[1]);
    int N = std::stoi(argv[2]);
    int K = std::stoi(argv[3]);

    float *A = new float[M * K];
    float *B = new float[N * K];
    float *C = new float[N * M];
    float *D = new float[N * M];
    for (int i = 0; i < M * K; i++) {
        A[i] = (rand() % 1000) * 0.01f;
        // A[i] = 1.0f;
    } 
    for (int i = 0; i < N * K; i++) {
        B[i] = (rand() % 1000) * 0.01f;
        // B[i] = 1.0f;
    }

    float *d_A, *d_B, *d_C, *d_D;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));
    hipMalloc(&d_D, M * N * sizeof(float));

    hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);

    void* args[] = {&d_A, &d_B, &d_C};
    dim3 dimBlock = {16, 16};
    dim3 dimGrid = {N  / dimBlock.x, M  / dimBlock.y};

    // 调用核函数
    hipError_t err;
    hipModuleLaunchKernel(kernel, std::stoi(argv[4]), 1, 1, std::stoi(argv[5]), 1, 1, std::stoi(argv[6]), 0, args, NULL);
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("设备同步失败: %s\n", hipGetErrorString(err));
        return 1;
    }

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    cublasMatMulTransA(handle, d_A, d_B, d_D, M, N, K, true, false);
    verify_kernel<<<dimGrid, dimBlock>>>(d_C, d_D, M, N);

    // 同步设备
    hipModuleUnload(module);
    hipCtxDestroy(context);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);

    delete[] A;
    delete[] B;
    delete[] C;
    delete[] D;

    return 0;
}