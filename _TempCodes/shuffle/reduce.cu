#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <float.h>

__global__ void reduce(float* input, float* output) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  float elem = input[(bid + (tid / 16) * 16) * 16 + tid % 16];
  for (int i=1; i<16; i*=2) {
    elem += __shfl_down_sync(0xffffffff, elem, i, 16);
  }
  output[(bid + (tid / 16) * 16) * 16 + tid % 16] = elem;
}

int main() {
  int device_count;
  hipError_t cuResult = hipInit(0);
  hipGetDeviceCount(&device_count);
  if (device_count == 0) {
    std::cerr << "没有找到可用的CUDA设备" << std::endl;
    return 1;
  }
  int device_id;
  hipGetDevice(&device_id);
  hipSetDevice(device_id);

  hipCtx_t context;
  cuResult = hipCtxCreate(&context, 0, device_id);
  if (cuResult != hipSuccess) {
    std::cerr << "Failed to create CUDA context: " << cuResult << std::endl;
    return 1;
  }

  // 加载.cubin模块
  hipModule_t module;
  hipError_t result = hipModuleLoad(&module, "/home/xiebaokang/projects/mlir/amendDeepGen/build/reduce.cubin");
  if (result != hipSuccess) {
    std::cerr << "Failed to load module: " << result << std::endl;
    return -1;
  }

  // 获取内核函数句柄
  hipFunction_t kernel;
  result = hipModuleGetFunction(&kernel, module, "reduce");
  if (result != hipSuccess) {
    std::cerr << "Failed to get kernel function" << std::endl;
    return -1;
  }


  // 分配主机内存
  int len = 32*16;
  float* h_input = new float[len];
  float* h_output = new float[len];

  // 初始化输入矩阵
  std::srand(1);
  for (int i = 0; i < len; ++i) {
    h_input[i] = (std::rand() % 1000) * 0.01f;
    // h_input[i] = 1.0f;
  }

  // 分配GPU内存
  float *d_input, *d_output;
  hipMalloc(&d_input, len * sizeof(float));
  hipMalloc(&d_output, len * sizeof(float));

  // 将数据从CPU复制到GPU
  hipMemcpy(d_input, h_input, len * sizeof(float), hipMemcpyHostToDevice);

  hipError_t err;
  void* args[] = {&d_input, &d_output};
  hipModuleLaunchKernel(kernel, 16, 1, 1, 32, 1, 1, 0, 0, args, NULL);
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
      printf("设备同步失败: %s\n", hipGetErrorString(err));
      return 1;
  }
  hipMemcpy(h_output, d_output, len * sizeof(float), hipMemcpyDeviceToHost);
  // display(h_O, len);
  for (int i=0; i<32; i++) {
    printf("%.7f ", h_output[i*16]);
  }
  printf("\n\n");

  dim3 gridSize1(16);  // bx, by, bz
  dim3 blockSize1(32);
  reduce<<<gridSize1, blockSize1>>>(d_input, d_output);
  hipMemcpy(h_output, d_output, len * sizeof(float), hipMemcpyDeviceToHost);
  // display(h_O, len);
  for (int i=0; i<32; i++) {
    printf("%.7f ", h_output[i*16]);
  }
  printf("\n\n");

  // 同步设备
  hipModuleUnload(module);
  hipCtxDestroy(context);

  // 释放主机内存
  delete[] h_input;
  delete[] h_output;
  hipFree(d_input);
  hipFree(d_output);

  return 0;
}    