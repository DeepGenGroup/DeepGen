#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "utils.h"
#include <cassert>

template <typename scalar_t, bool IS_NEOX>
inline __device__ void apply_rotary_embedding(scalar_t* __restrict__ arr, const scalar_t* __restrict__ cos_ptr, const scalar_t* __restrict__ sin_ptr, int rot_offset, int embed_dim) {
  int x_index, y_index;
  scalar_t cos, sin;
  if (IS_NEOX) {
    // GPT-NeoX style rotary embedding.
    x_index = rot_offset;
    y_index = embed_dim + rot_offset;
    cos = __ldg(cos_ptr + x_index);
    sin = __ldg(sin_ptr + x_index);
  } else {
    // GPT-J style rotary embedding.
    x_index = 2 * rot_offset;
    y_index = 2 * rot_offset + 1;
    cos = __ldg(cos_ptr + x_index / 2);
    sin = __ldg(sin_ptr + x_index / 2);
  }

  const scalar_t x = arr[x_index];
  const scalar_t y = arr[y_index];
  arr[x_index] = x * cos - y * sin;
  arr[y_index] = y * cos + x * sin;
}

template <typename scalar_t, bool IS_NEOX>
__global__ void rotary_embedding_kernel(const int32_t* __restrict__ positions,       // [batch_size, seq_len] or [num_tokens]
                                        scalar_t* __restrict__ query,                // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
                                        scalar_t* __restrict__ key,                  // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
                                        const scalar_t* __restrict__ cos_sin_cache,  // [max_position, 2, rot_dim // 2]
                                        const int rot_dim, const int64_t query_stride, const int64_t key_stride, const int num_heads, const int num_kv_heads, const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const scalar_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const scalar_t* cos_ptr = cache_ptr;
  const scalar_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<scalar_t, IS_NEOX>(query + token_head, cos_ptr, sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<scalar_t, IS_NEOX>(key + token_head, cos_ptr, sin_ptr, rot_offset, embed_dim);
  }
}


template <typename scalar_t, int HEAD_DIM>
__global__ void rotary_embedding_online_kernel(
  const int32_t* __restrict__ positions,
  scalar_t* __restrict__ query,
  scalar_t* __restrict__ key,
  const float rope_theta,
  const int64_t query_stride,
  const int64_t key_stride,
  const int head_num,
  const int kv_head_num
) { 

  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];

  constexpr int EMBED_DIM = HEAD_DIM / 2;
  // precompute
  __shared__ float freq_cos[EMBED_DIM];
  __shared__ float freq_sin[EMBED_DIM];
  for (int i = threadIdx.x; i < EMBED_DIM; i += blockDim.x) {
    float freq = float(pos) / __powf(rope_theta, float(2 * i) / float(HEAD_DIM));
    __sincosf(freq, &freq_sin[i], &freq_cos[i]);
  }
  __syncthreads();

  const int nq = head_num * EMBED_DIM;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / EMBED_DIM;
    const int64_t token_head = token_idx * query_stride + head_idx * HEAD_DIM;
    int x_index = i % EMBED_DIM;
    int y_index = x_index + EMBED_DIM;
    const scalar_t x = query[token_head + x_index];
    const scalar_t y = query[token_head + y_index];
    query[token_head + x_index] = x * freq_cos[x_index] - y * freq_sin[x_index];
    query[token_head + y_index] = y * freq_cos[x_index] + x * freq_sin[x_index];
  }

  const int nk = kv_head_num * EMBED_DIM;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / EMBED_DIM;
    const int64_t token_head = token_idx * key_stride + head_idx * HEAD_DIM;
    const int rot_offset = i % EMBED_DIM;
    int x_index = i % EMBED_DIM;
    int y_index = x_index + EMBED_DIM;
    const scalar_t x = key[token_head + x_index];
    const scalar_t y = key[token_head + y_index];
    key[token_head + x_index] = x * freq_cos[x_index] - y * freq_sin[x_index];
    key[token_head + y_index] = y * freq_cos[x_index] + x * freq_sin[x_index];
  }
}


template <typename scalar_t, int HEAD_DIM>
__global__ void rotary_embedding_single_online_kernel(
  const int32_t* __restrict__ positions,
  scalar_t* __restrict__ arr,
  const float rope_theta,
  const int64_t stride,
  const int head_num
) { 

  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];

  constexpr int EMBED_DIM = HEAD_DIM / 2;
  // precompute
  __shared__ float freq_cos[EMBED_DIM];
  __shared__ float freq_sin[EMBED_DIM];
  for (int i = threadIdx.x; i < EMBED_DIM; i += blockDim.x) {
    float freq = float(pos) / __powf(rope_theta, float(2 * i) / float(HEAD_DIM));
    __sincosf(freq, &freq_sin[i], &freq_cos[i]);
  }
  __syncthreads();

  const int n = head_num * EMBED_DIM;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int head_idx = i / EMBED_DIM;
    const int64_t token_head = token_idx * stride + head_idx * HEAD_DIM;
    int x_index = i % EMBED_DIM;
    int y_index = x_index + EMBED_DIM;
    const scalar_t x = arr[token_head + x_index];
    const scalar_t y = arr[token_head + y_index];
    arr[token_head + x_index] = x * freq_cos[x_index] - y * freq_sin[x_index];
    arr[token_head + y_index] = y * freq_cos[x_index] + x * freq_sin[x_index];
  }
}


void rotary_embedding(torch::Tensor& positions,  // [batch_size, seq_len] or [num_tokens]
                      torch::Tensor& query,      // [batch_size, seq_len, num_heads * head_size] or [num_tokens, num_heads * head_size]
                      torch::Tensor& key,        // [batch_size, seq_len, num_kv_heads * head_size] or [num_tokens, num_kv_heads * head_size]
                      int head_size,
                      torch::Tensor& cos_sin_cache,  // [max_position, rot_dim]
                      bool is_neox) {
  int64_t num_tokens = query.numel() / query.size(-1);
  int rot_dim = cos_sin_cache.size(1);
  int num_heads = query.size(-1) / head_size;
  int num_kv_heads = key.size(-1) / head_size;
  int64_t query_stride = query.stride(-2);
  int64_t key_stride = key.stride(-2);

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * rot_dim / 2, 512));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  DISPATCH_FLOATING_TYPES(query.scalar_type(), "rotary_embedding", [&] {
    if (is_neox) {
      rotary_embedding_kernel<scalar_t, true><<<grid, block, 0, stream>>>(positions.data_ptr<int32_t>(), query.data_ptr<scalar_t>(), key.data_ptr<scalar_t>(), cos_sin_cache.data_ptr<scalar_t>(),
                                                                          rot_dim, query_stride, key_stride, num_heads, num_kv_heads, head_size);
    } else {
      rotary_embedding_kernel<scalar_t, false><<<grid, block, 0, stream>>>(positions.data_ptr<int32_t>(), query.data_ptr<scalar_t>(), key.data_ptr<scalar_t>(), cos_sin_cache.data_ptr<scalar_t>(),
                                                                           rot_dim, query_stride, key_stride, num_heads, num_kv_heads, head_size);
    }
  });
}


void rotary_embedding_online(torch::Tensor& positions,  // [num_tokens]
                             torch::Tensor& query,      // [num_tokens, num_heads * head_size]
                             torch::Tensor& key,        // [num_tokens, num_kv_heads * head_size]
                             int head_dim,
                             float rope_theta
) {
  int64_t token_num = query.numel() / query.size(-1);
  int head_num = query.size(-1) / head_dim;
  int kv_head_num = key.size(-1) / head_dim;
  int64_t query_stride = query.stride(-2);
  int64_t key_stride = key.stride(-2);

  dim3 grid(token_num);
  dim3 block(std::min(head_num * head_dim / 2, 512));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  assert(head_dim == 128);
  DISPATCH_FLOATING_TYPES(query.scalar_type(), "rotary_embedding_online", [&] {
    rotary_embedding_online_kernel<scalar_t, 128><<<grid, block, 0, stream>>>(positions.data_ptr<int32_t>(), query.data_ptr<scalar_t>(), key.data_ptr<scalar_t>(), rope_theta, query_stride, key_stride, head_num, kv_head_num);
  });
}

void rotary_embedding_single_online(torch::Tensor& positions,  // [num_tokens]
                                    torch::Tensor& arr,      // [num_tokens, num_heads * head_size]
                                    int head_dim,
                                    float rope_theta
) {
  int64_t token_num = arr.numel() / arr.size(-1);
  int head_num = arr.size(-1) / head_dim;
  int64_t stride = arr.stride(-2);

  dim3 grid(token_num);
  dim3 block(std::min(head_num * head_dim / 2, 512));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  assert(head_dim == 128);
  DISPATCH_FLOATING_TYPES(arr.scalar_type(), "rotary_embedding_single_online", [&] {
    rotary_embedding_single_online_kernel<scalar_t, 128><<<grid, block, 0, stream>>>(positions.data_ptr<int32_t>(), arr.data_ptr<scalar_t>(), rope_theta, stride, head_num);
  });
}

// TODO: remove redundant code